#include "hip/hip_runtime.h"
#include "CudaCommon.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cfloat>
#include <vector>
#include "Sphere.h"
#include "HitableList.h"
#include "Ray.h"
#include "Camera.h"
#include "Material.h"

const int MAX_DEPTH = 50;

CALLABLE_FUNC Vector3 color(const Ray& r, Hitable* world, unsigned int *seed0, unsigned int *seed1)
{
    Vector3 accumCol(1, 1, 1);
    Vector3 attenuation(0, 0, 0);

    Ray currentRay(r);

    for (int depth = 0; depth < MAX_DEPTH; depth++)
    {
        HitRecord rec;
        if (world->hit(currentRay, 0.001f, FLT_MAX, rec))
        {
            Ray scattered;
            if (rec.material->scatter(currentRay, rec, attenuation, scattered, seed0, seed1))
            {
                accumCol *= attenuation;
                currentRay = scattered;
            }
            else
            {
                accumCol = Vector3(0.0f, 0.0f, 0.0f);
                break;
            }
        }
        else
        {
            Vector3 unit_dir = unit_vector(r.direction());
            float t = 0.5f * (unit_dir.y() + 1.0f);

            attenuation = (1.0f - t) * Vector3(1.0f, 1.0f, 1.0f) + t * Vector3(0.5f, 0.7f, 1.0f);
            accumCol *= attenuation;
            break;
        }
    }
    return accumCol;
}

const int nx = 128*4;
const int ny = 128*4;

__device__ Camera g_cam;

__global__ void render_kernel(float3* pOutImage, Hitable** world, int nx, int ny, int ns)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nx || y >= ny) return;

    unsigned int i = (ny - y - 1) * nx + x; // index of current pixel (calculated using thread index)

    unsigned int seed0 = x;  // seeds for random number generator
    unsigned int seed1 = y;

    Vector3 accumCol(0, 0, 0);
    for (int s = 0; s < ns; s++)
    {
        float u = (x + rand(&seed0, &seed1)) / float(nx);
        float v = (y + rand(&seed0, &seed1)) / float(ny);
        Ray r = g_cam.getRay(u, v, &seed0, &seed1);
        accumCol += color(r, *world, &seed0, &seed1);
    }
    accumCol /= float(ns);

    pOutImage[i] = make_float3(sqrtf(accumCol[0]), sqrtf(accumCol[1]), sqrtf(accumCol[2]));
}

__global__ void allocate_world_kernel(Hitable** world, float aspect)
{
    int i = 0;
    Hitable** list = new Hitable*[4];
    list[i++] = new Sphere(Vector3(0.0f, 0.0f, -1.0f), 0.5f, new Lambertian(new ConstantTexture(Vector3(0.1, 0.2, 0.5))));
    list[i++] = new Sphere(Vector3(0.0f, -100.5f, -1.0f), 100.0f, new Lambertian(new ConstantTexture(Vector3(0.8, 0.8, 0.0))));
    list[i++] = new Sphere(Vector3(1, 0, -1), 0.5, new Metal(Vector3(0.8, 0.6, 0.2), 0.3));
    list[i++] = new Sphere(Vector3(-1, 0, -1), 0.5, new Dielectric(1.5));

    *world = new HitableList(i, list);

    g_cam = Camera(Vector3(-2, 2, 1), Vector3(0, 0, -1), Vector3(0, 1, 0), 90, aspect, 0.0f, 10.0f);
}

/*
__global__ void cornell_box_kernel(Hitable **world, float aspect)
{
    int i = 0;
    Hitable **list = new Hitable*[8];
    Material *red = new Lambertian( new ConstantTexture(Vector3(0.65f, 0.05f, 0.05f)) );
    Material *white = new Lambertian( new ConstantTexture(Vector3(0.73, 0.73, 0.73)) );
    Material *green = new Lambertian( new ConstantTexture(Vector3(0.12, 0.45, 0.15)) );
    Material *light = new DiffuseLight( new ConstantTexture(Vector3(15, 15, 15)) );
    Material* aluminum = new Metal(Vector3(0.8, 0.85, 0.88), 0.0);

    list[i++] = new FlipNormals(new YZRectangle(0, 555, 0, 555, 555, green));
    list[i++] = new YZRectangle(0, 555, 0, 555, 0, red);
    list[i++] = new FlipNormals(new XZRectangle(213, 343, 227, 332, 554, light));
    list[i++] = new FlipNormals(new XZRectangle(0, 555, 0, 555, 555, white));
    list[i++] = new XZRectangle(0, 555, 0, 555, 0, white);
    list[i++] = new FlipNormals(new XYRectangle(0, 555, 0, 555, 555, white));
    //Material *glass = new Dielectric(1.5);
    //list[i++] = new Sphere(Vector3(190, 90, 190),90 , glass);
    list[i++] = new Translate(new RotateY(new Box(Vector3(0, 0, 0), Vector3(165, 165, 165), white), -18), Vector3(130, 0, 65));
    list[i++] = new Translate(new RotateY(new Box(Vector3(0, 0, 0), Vector3(165, 330, 165), aluminum),  90), Vector3(265,0,295));
    //list[i++] = new Translate(new Box(Vector3(0, 0, 0), Vector3(165, 330, 165), aluminum), Vector3(265,0,295));
    *world = new HitableList(list,i);
    Vector3 lookfrom(278, 278, -800);
    Vector3 lookat(278,278,0);
    float dist_to_focus = 10.0f;
    float aperture = 0.0f;
    float vfov = 40.0f;
    g_cam = Camera(lookfrom, lookat, Vector3(0,1,0),
                      vfov, aspect, aperture, dist_to_focus, 0.0, 1.0);
}
*/
int main()
{
    int ns = 1000;

    /*
    std::vector<Hitable*> list;
    list.push_back(new Sphere(Vector3(0.0f, 0.0f, -1.0f), 0.5f, new Lambertian(Vector3(0.1, 0.2, 0.5))));
    list.push_back(new Sphere(Vector3(0.0f, -100.5f, -1.0f), 100.0f, new Lambertian(Vector3(0.8, 0.8, 0.0))));
    list.push_back(new Sphere(Vector3(1, 0, -1), 0.5, new Metal(Vector3(0.8, 0.6, 0.2), 0.3)));
    list.push_back(new Sphere(Vector3(-1, 0, -1), 0.5, new Dielectric(1.5)));
    //list.push_back(new Sphere(Vector3(-1,0,-1), -0.45, new Dielectric(1.5)));

    //double R = cos(M_PI/4);
    //list.push_back(new Sphere(Vector3(-R,0,-1), R, new Lambertian(Vector3(0,0,1))));
    //list.push_back(new Sphere(Vector3(R,0,-1), R, new Lambertian(Vector3(1,0,0))));

    Hitable* world = new HitableList(list.size(), list.data());

    unsigned int seed0;
    unsigned int seed1;
    //Camera cam(90, double(nx)/ny);
    Camera cam(Vector3(-2, 2, 1), Vector3(0, 0, -1), Vector3(0, 1, 0), 90, float(nx)/float(ny), 0.0f, 10.0f);
    for (int j = ny-1; j>=0; j--)
    {
        seed0 = j;
        for (int i = 0; i<nx; i++)
        {
            seed1 = i;
            Vector3 col(0, 0, 0);
            for (int s = 0; s<ns; s++)
            {
                float u = (i+drand48())/float(nx);
                float v = (j+drand48())/float(ny);
                Ray r = cam.getRay(u, v, &seed0, &seed1);
                col += color(r, world, 0, &seed0, &seed1);
            }
            col /= float(ns);
            col = Vector3(sqrtf(col[0]), sqrtf(col[1]), sqrtf(col[2]));
            int ir = int(255.99f*col[0]);
            int ig = int(255.99f*col[1]);
            int ib = int(255.99f*col[2]);

            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    */

    const float aspect = float(nx)/float(ny);

    float3* pOutImage = NULL;
    hipMalloc(&pOutImage, nx * ny * sizeof(float3));

    Hitable** world = NULL;
    hipMalloc(&world, sizeof(Hitable**));

    std::cerr << "Allocating world...";
    allocate_world_kernel<<<1, 1>>>(world, aspect);
    hipError_t err = hipDeviceSynchronize();
    std::cerr << "done" << std::endl;
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate world in GPU memory.  Error: " << hipGetErrorName(err) << " Desc: " << hipGetErrorString(err) << std::endl;
        return EXIT_FAILURE;
    }

    dim3 block(8, 8, 1);
    dim3 grid(IDIVUP(nx, block.x), IDIVUP(ny, block.y), 1);
    std::cerr << "Rendering world...";
    render_kernel<<<grid, block>>>(pOutImage, world, nx, ny, ns);
    err = hipDeviceSynchronize();
    std::cerr << "done" << std::endl;
    if (err != hipSuccess)
    {
        std::cerr << "Failed to render on GPU.  Error: " << hipGetErrorName(err) << " Desc: " << hipGetErrorString(err) << std::endl;
        return EXIT_FAILURE;
    }

    float3* pTemp = new float3[nx * ny];
    hipMemcpy(pTemp, pOutImage, nx*ny*sizeof(float3), hipMemcpyDeviceToHost);

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int i = 0; i < nx * ny; i++)
    {
        int ir = int(255.99f * pTemp[i].x);
        int ig = int(255.99f * pTemp[i].y);
        int ib = int(255.99f * pTemp[i].z);
        std::cout << ir << " " << ig << " " << ib << "\n";
    }

    std::cerr << "Done." << std::endl;
    hipFree(pOutImage);
    delete[] pTemp;

    return EXIT_SUCCESS;
}
